#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define XINICIAL 0
#define XFINAL 10
#define PASOS 1000000000

int *res;  // host data
int *c;  // results

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}

float calcularFuncion(float x)
{
    return x * x;
}

//GPU kernel 
__global__
void vecAdd(int *res,int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

//CPU function
void integral_h(float *res, int N){
    float incremento = ((float)XFINAL - (float)XINICIAL) / (float)PASOS;
    float x = XINICIAL;
    for (int i = 0; i < PASOS; i++)
    {
        res += incremento * (calcularFuncion(x) + calcularFuncion(x + incremento)) / (float)2;
        x += incremento;
    }
}

int main(int argc,char **argv)
{
    printf("Begin \n");
    //Iterations
    int n=1000000000;
    //Number of blocks
    int nBytes = n*sizeof(float);
    //Block size and number
    int block_size, block_no;

    //memory allocation	
    res = (float *) malloc(nBytes);
    c = (float *) malloc(nBytes);

    int *a_d,*b_d,*c_d;
    block_size = 250; //threads per block
    block_no = n/block_size;
        
    //Work definition
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(block_no, 1, 1);

    // Data filling
    for(int i=0;i<n;i++)
    a[i]=i,b[i]=i;


    printf("Allocating device memory on host..\n");
   //GPU memory allocation
    hipMalloc((void **) &a_d, n*sizeof(float));
    hipMalloc((void **) &b_d, n*sizeof(float));
    hipMalloc((void **) &c_d, n*sizeof(float));

    printf("Copying to device..\n");
    hipMemcpy(a_d, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(float), hipMemcpyHostToDevice);

    clock_t start_d=clock();
    printf("Doing GPU Vector add\n");
    vecAdd<<<block_no,block_size>>>(a_d, b_d, c_d, n);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    clock_t end_d = clock();
    

    printf("Doing CPU Vector add\n");
    clock_t start_h = clock();
    vecAdd_h(a, b, c2, n);
    clock_t end_h = clock();
	
    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    hipMemcpy(c, c_d, n*sizeof(int), hipMemcpyDeviceToHost);
    printf("n = %d \t GPU time = %fs \t CPU time = %fs\n", n, time_d, time_h);

    //Free GPU memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}
